#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <map>
#include <chrono>

struct Point {
  double x; // meters
  double y; // meters
  double z; // meters
  double vx; // meters/second
  double vy; // meters/second
  double vz; // meters/second
  double mass; // kg
  double fx; // N - reset every iteration
  double fy; // N
  double fz; // N
};

struct Spring {
  double k; // N/m
  int p1; // Index of first point
  int p2; // Index of second point
  double l0; // meters
};

void genPointsAndSprings(
	std::vector<Point> &points,
	std::vector<Spring> &springs,
	std::vector<std::vector<Spring>> &pointSprings);

const double staticFriction = 0.5;
const double kineticFriction = 0.3;
const double dt = 0.0000005;
const double dampening = 1 - (dt * 1000);
const double gravity = -9.81;
const double kSpring = 10000.0;
const double kGround = 100000.0;
const double kOscillationFrequency = 0;//10000;//100000
const double kDropHeight = 0.2;
const int pointsPerSide = 10;

__global__ void device_add(int *a, int *b, int *c) {
     c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

/*int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = N * sizeof(int);
 
    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size);
    fill_array(a);
    b = (int *)malloc(size);
    fill_array(b);
    c = (int *)malloc(size);
 
    // Alloc space for device copies of vector (a, b, c)
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));
 
    // Copy from host to device
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);
 
    device_add<<<1,N>>>(d_a,d_b,d_c);
 
    // Copy result back to host
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
 
    print_output(a,b,c);
    free(a); free(b); free(c);
 
    //free gpu memory
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
 
    return 0;
}*/

int main() {
    std::vector<Point> points;
    std::vector<Spring> springs;
    std::vector<std::vector<Spring>> pointSprings;

    genPointsAndSprings(points, springs, pointSprings);
    double t = 0;
    // 60 fps - 0.000166
    double limit = 1.0;
  
  	int numSprings = (int)springs.size();
  	printf("%f, %f\n", t, points[0].mass);
    printf("num springs evaluated: %lld\n", long long int(limit / dt * numSprings));
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    while (t < limit) {
        double adjust = 1 + sin(t * kOscillationFrequency) * 0.1;
        for (int i = 0; i < springs.size(); i++) {
            Spring l = springs[i];

            int p1index = l.p1;
            int p2index = l.p2;
            Point p1 = points[p1index];
            Point p2 = points[p2index];

            double p1x = p1.x;
            double p1y = p1.y;
            double p1z = p1.z;
            double p2x = p2.x;
            double p2y = p2.y;
            double p2z = p2.z;
            double dist = sqrt(pow(p1x - p2x, 2) + pow(p1y - p2y, 2) + pow(p1z - p2z, 2));

            // negative if repelling, positive if attracting
            double f = l.k * (dist - (l.l0 * adjust));
            // distribute force across the axes
            double dx = f * (p1x - p2x) / dist;
            points[p1index].fx -= dx;
            points[p2index].fx += dx;

            double dy = f * (p1y - p2y) / dist;
            points[p1index].fy -= dy;
            points[p2index].fy += dy;

            double dz = f * (p1z - p2z) / dist;
            points[p1index].fz -= dz;
            points[p2index].fz += dz;
        }
        for (int i = 0; i < points.size(); i++) {
            Point p = points[i];
        
            double mass = p.mass;
            double fy = p.fy + gravity * mass;
            double fx = p.fx;
            double fz = p.fz;
            double y = p.y;
            double vx = p.vx;
            double vy = p.vy;
            double vz = p.vz;

            if (y <= 0) {
                double fh = sqrt(pow(fx, 2) + pow(fz, 2));
                double fyfric = abs(fy * staticFriction);
                if (fh < fyfric) {
                    fx = 0;
                    fz = 0;
                } else {
                    double fykinetic = abs(fy * kineticFriction);
                    fx = fx - fx / fh * fykinetic;
                    fz = fz - fz / fh * fykinetic;
                }
                fy += -kGround * y;
            }
            double ax = fx / mass;
            double ay = fy / mass;
            double az = fz / mass;
            // reset the force cache
            p.fx = 0;
            p.fy = 0;
            p.fz = 0;
            vx = (ax * dt + vx) * dampening;
            p.vx = vx;
            vy = (ay * dt + vy) * dampening;
            p.vy = vy;
            vz = (az * dt + vz) * dampening;
            p.vz = vz;
            p.x += vx;
            p.y += vy;
            p.z += vz;
            points[i] = p;
        }
        t += dt;
    }

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin);
    std::cout << "Time difference = " << ms.count() / 1000.0 << "[s]" << std::endl;
    printf("%f, %f\n", t, points[0].fy);

    return 0;
}

void genPointsAndSprings(
	std::vector<Point> &points,
	std::vector<Spring> &springs,
	std::vector<std::vector<Spring>> &pointSprings) {
    std::map<int, std::map<int, std::map<int, Point>>> cache;

    for (int x = 0; x < pointsPerSide; x++) {
        for (int y = 0; y < pointsPerSide; y++) {
            for (int z = 0; z < pointsPerSide; z++) {
                // (0,0,0) or (0.1,0.1,0.1) and all combinations
                Point p = {x / 10.0, kDropHeight + y / 10.0, z / 10.0, 0, 0, 0, 0.1, 0, 0, 0};
                points.push_back(p);
                if (cache.count(x) == 0) {
                    cache[x] = {};
                }
                if (cache[x].count(y) == 0) {
                    cache[x][y] = {};
                }
                cache[x][y][z] = p;
                pointSprings.push_back({});
            }
        }
    }
    // Create the springs
    for (int x = 0; x < pointsPerSide; x++) {
        for (int y = 0; y < pointsPerSide; y++) {
            for (int z = 0; z < pointsPerSide; z++) {
                Point p1 = cache[x][y][z];
                int p1index = z + pointsPerSide * y + pointsPerSide * pointsPerSide * x;
                for (int x1 = x; x1 < x + 2; x1++) {
                    if (x1 == pointsPerSide) {
                        continue;
                    }
                    for (int y1 = y; y1 < y + 2; y1++) {
                        if (y1 == pointsPerSide) {
                            continue;
                        }
                        for (int z1 = z; z1 < z + 2; z1++) {
                            if (z1 == pointsPerSide || (x1 == x && y1 == y && z1 == z)) {
                                continue;
                            }
                            Point p2 = cache[x1][y1][z1];
                            int p2index = z1 + pointsPerSide * y1 + pointsPerSide * pointsPerSide * x1;
                            double length = sqrt(pow(p1.x - p2.x, 2) + pow(p1.y - p2.y, 2) + pow(p1.z - p2.z, 2));
                            Spring s = {kSpring, p1index, p2index, length};
                            springs.push_back(s);
                            pointSprings[p1index].push_back(s);
							pointSprings[p2index].push_back(s);
                        }
                    }
                }
            }
        }
    }
}
