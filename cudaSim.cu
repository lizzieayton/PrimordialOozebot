#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <map>
#include <chrono>

// Usage: nvcc -O2 cudaSim.cu -o cudaSim -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Tools\MSVC\14.27.29110\bin\Hostx64\x64"

struct Point {
  double x; // meters
  double y; // meters
  double z; // meters
  double vx; // meters/second
  double vy; // meters/second
  double vz; // meters/second
  double mass; // kg
  int numSprings; // Int - hack for CUDA ease
};

struct Spring {
  double k; // N/m
  int p1; // Index of first point
  int p2; // Index of second point
  double l0; // meters
  double dx; // caching for CUDA ease
  double dy; // caching for CUDA ease
  double dz; // caching for CUDA ease
};

void genPointsAndSprings(
	std::vector<Point> &points,
	std::vector<Spring> &springs,
	std::vector<std::vector<Spring>> &pointSprings);

#define staticFriction 0.5
#define kineticFriction 0.3
#define dt 0.0000005
#define dampening 1 - (0.0000005 * 1000)
#define gravity -9.81
#define kSpring 10000.0
#define kGround 100000.0
const double kOscillationFrequency = 0;
const double kDropHeight = 0.2;
const int pointsPerSide = 20;

__global__ void update_spring(Point *points, Spring *springs, double adjust) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    Point p1 = points[s[i].p1];
    Point p2 = points[s[i].p2];

    double p1x = p1.x;
    double p1y = p1.y;
    double p1z = p1.z;
    double p2x = p2.x;
    double p2y = p2.y;
    double p2z = p2.z;
    double dist = sqrt(pow(p1x - p2x, 2) + pow(p1y - p2y, 2) + pow(p1z - p2z, 2));

    // negative if repelling, positive if attracting
    double f = l.k * (dist - (l.l0 * adjust));
    // distribute force across the axes
    double dx = f * (p1x - p2x) / dist;
    double dy = f * (p1y - p2y) / dist;
    double dz = f * (p1z - p2z) / dist;

    springs
    s[i].dx = dx;
    s[i].dy = dy;
    s[i].dz = dz;
}

__global__ void update_point(Point *points, Spring *springs, int **pointsToSprings) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d, %d, %d, %d, %d, %d\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z);

    Point p = points[i];
    double fy = gravity * mass;
    double fx = 0;
    double fz = 0;
    int *pToS = pointSprings[i]

    for (int j = 0; j < p.numSprings; j++) {
        Spring s = pToS[j];
        if (s.p1 == i) {
            fx -= s.dx;
            fy -= s.dy;
            fz -= s.dz;
        } else {
            fx += s.dx;
            fy += s.dy;
            fz += s.dz;
        }
    }
        
    double mass = p.mass;
    double y = p.y;
    double vx = p.vx;
    double vy = p.vy;
    double vz = p.vz;

    if (y <= 0) {
        double fh = sqrt(pow(fx, 2) + pow(fz, 2));
        double fyfric = abs(fy * staticFriction);
        if (fh < fyfric) {
            fx = 0;
            fz = 0;
        } else {
            double fykinetic = abs(fy * kineticFriction);
            fx = fx - fx / fh * fykinetic;
            fz = fz - fz / fh * fykinetic;
        }
        fy += -kGround * y;
    }
    double ax = fx / mass;
    double ay = fy / mass;
    double az = fz / mass;
    // reset the force cache
    vx = (ax * dt + vx) * dampening;
    p.vx = vx;
    vy = (ay * dt + vy) * dampening;
    p.vy = vy;
    vz = (az * dt + vz) * dampening;
    p.vz = vz;
    p.x += vx;
    p.y += vy;
    p.z += vz;
    points[i] = p;
}

int main() {
    std::vector<Point> points;
    std::vector<Spring> springs;
    std::vector<std::vector<Spring>> pointSprings;

    genPointsAndSprings(points, springs, pointSprings);
    Point *p_d;
    Spring *p_d;
    int **ps_d;
    std::vector<int *> psd;
    hipMalloc(&p_d, points.size() * sizeof(Point));
    hipMemcpy(p_d, &points[0], points.size() * sizeof(Point), hipMemcpyHostToDevice);
    hipMalloc(&p_d, springs.size() * sizeof(Spring));
    hipMemcpy(p_d, &springs[0], points.size() * sizeof(Spring), hipMemcpyHostToDevice);
    hipMalloc(&ps_d, points.size() * sizeof(int *));
    for (int i = 0; i < points.size(); i++) {
    	Spring *s_d;
    	psd.push_back(s_d);
    	hipMalloc(&s_d, pointSprings[i].size() * sizeof(int));
    	hipMemcpy(s_d, &pointSprings[i], pointSprings[i].size() * sizeof(int), hipMemcpyHostToDevice);
    }

    double t = 0;
    // 60 fps - 0.000166
    double limit = 0.1;
    int ppsSquare = pointsPerSide * pointsPerSide;
  
  	int numSprings = (int)springs.size();

    if (numSprings % 1000 != 0) {
        pringf("Whoa, issue with num springs\n");
    }
    int springThreads = 1000;
    int springBlocks = numSprings / 1000;
    printf("num springs evaluated: %lld\n", long long int(limit / dt * numSprings));
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    while (t < limit) {
        double adjust = 1 + sin(t * kOscillationFrequency) * 0.1;
        
        update_spring<<<ppsSquare, pointsPerSide>>>(p_d, s_d, adjust);
        hipDeviceSynchronize();
        update_point<<<springBlocks, springThreads>>>(p_d, p_d, ps_d);
        hipDeviceSynchronize();
 
        t += dt;
    }

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin);
    std::cout << "Time difference = " << ms.count() / 1000.0 << "[s]" << std::endl;

    Point *ps = (Point *)malloc(points.size() * sizeof(Point));
    hipMemcpy(ps, p_d, points.size() * sizeof(Point), hipMemcpyDeviceToHost);
    
    hipFree(p_d);
    for (int i = 0; i < points.size(); i++) {
    	hipFree(psd[i]);
    }
    hipFree(ps_d);
    free(ps);

    return 0;
}

void genPointsAndSprings(
	std::vector<Point> &points,
	std::vector<Spring> &springs,
	std::vector<std::vector<Spring>> &pointSprings) {
    std::map<int, std::map<int, std::map<int, Point>>> cache;

    for (int x = 0; x < pointsPerSide; x++) {
        for (int y = 0; y < pointsPerSide; y++) {
            for (int z = 0; z < pointsPerSide; z++) {
                // (0,0,0) or (0.1,0.1,0.1) and all combinations
                Point p = {x / 10.0, kDropHeight + y / 10.0, z / 10.0, 0, 0, 0, 0.1, 0, 0, 0, 0};
                points.push_back(p);
                if (cache.count(x) == 0) {
                    cache[x] = {};
                }
                if (cache[x].count(y) == 0) {
                    cache[x][y] = {};
                }
                cache[x][y][z] = p;
                pointSprings.push_back({});
            }
        }
    }
    // Create the springs
    for (int x = 0; x < pointsPerSide; x++) {
        for (int y = 0; y < pointsPerSide; y++) {
            for (int z = 0; z < pointsPerSide; z++) {
                Point p1 = cache[x][y][z];
                int p1index = z + pointsPerSide * y + pointsPerSide * pointsPerSide * x;
                for (int x1 = x; x1 < x + 2; x1++) {
                    if (x1 == pointsPerSide) {
                        continue;
                    }
                    for (int y1 = y; y1 < y + 2; y1++) {
                        if (y1 == pointsPerSide) {
                            continue;
                        }
                        for (int z1 = z; z1 < z + 2; z1++) {
                            if (z1 == pointsPerSide || (x1 == x && y1 == y && z1 == z)) {
                                continue;
                            }
                            Point p2 = cache[x1][y1][z1];
                            int p2index = z1 + pointsPerSide * y1 + pointsPerSide * pointsPerSide * x1;
                            double length = sqrt(pow(p1.x - p2.x, 2) + pow(p1.y - p2.y, 2) + pow(p1.z - p2.z, 2));
                            Spring s = {kSpring, p1index, p2index, length};
                            springs.push_back(s);
                            pointSprings[p1index].push_back(s);
							pointSprings[p2index].push_back(s);
							p2.numSprings += 1;
							p1.numSprings += 1;
                        }
                    }
                }
            }
        }
    }
}
