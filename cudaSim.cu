#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <map>
#include <chrono>

// Usage: nvcc -O2 cudaSim.cu -o cudaSim -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Tools\MSVC\14.27.29110\bin\Hostx64\x64"

struct Point {
  double x; // meters
  double y; // meters
  double z; // meters
  double vx; // meters/second
  double vy; // meters/second
  double vz; // meters/second
  double mass; // kg
  double fx; // N - reset every iteration
  double fy; // N
  double fz; // N
  int numSprings; // Int - hack for CUDA ease
};

struct Spring {
  double k; // N/m
  int p1; // Index of first point
  int p2; // Index of second point
  double l0; // meters
};

void genPointsAndSprings(
	std::vector<Point> &points,
	std::vector<Spring> &springs,
	std::vector<std::vector<Spring>> &pointSprings);

#define staticFriction 0.5
#define kineticFriction 0.3
#define dt 0.0000005
#define dampening 1 - (0.0000005 * 1000)
#define gravity -9.81
#define kSpring 10000.0
#define kGround 100000.0
const double kOscillationFrequency = 0;
const double kDropHeight = 0.2;
const int pointsPerSide = 20;

__global__ void update_point(Point *points, Spring **pointsToSprings, double adjust) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	Point p1 = points[i];

	for (int j = 0; j < p1.numSprings; j++) {
    	Spring l = pointsToSprings[i][j];
    	int p2index = l.p2;
    	if (i == p2index) {
    		p2index = l.p1;
    	}

        Point p2 = points[p2index];

        double p1x = p1.x;
        double p1y = p1.y;
        double p1z = p1.z;
        double p2x = p2.x;
        double p2y = p2.y;
        double p2z = p2.z;
        double dist = sqrt(pow(p1x - p2x, 2) + pow(p1y - p2y, 2) + pow(p1z - p2z, 2));

        // negative if repelling, positive if attracting
        double f = l.k * (dist - (l.l0 * adjust));
        // distribute force across the axes
        double dx = f * (p1x - p2x) / dist;
        double dy = f * (p1y - p2y) / dist;
        double dz = f * (p1z - p2z) / dist;

        p1.fx -= dx;
        p1.fy -= dy;
        p1.fz -= dz;
    }
    Point p = p1;
        
    double mass = p.mass;
    double fy = p.fy + gravity * mass;
    double fx = p.fx;
    double fz = p.fz;
    double y = p.y;
    double vx = p.vx;
    double vy = p.vy;
    double vz = p.vz;

    if (y <= 0) {
        double fh = sqrt(pow(fx, 2) + pow(fz, 2));
        double fyfric = abs(fy * staticFriction);
        if (fh < fyfric) {
            fx = 0;
            fz = 0;
        } else {
            double fykinetic = abs(fy * kineticFriction);
            fx = fx - fx / fh * fykinetic;
            fz = fz - fz / fh * fykinetic;
        }
        fy += -kGround * y;
    }
    double ax = fx / mass;
    double ay = fy / mass;
    double az = fz / mass;
    // reset the force cache
    p.fx = 0;
    p.fy = 0;
    p.fz = 0;
    vx = (ax * dt + vx) * dampening;
    p.vx = vx;
    vy = (ay * dt + vy) * dampening;
    p.vy = vy;
    vz = (az * dt + vz) * dampening;
    p.vz = vz;
    p.x += vx;
    p.y += vy;
    p.z += vz;
    points[i] = p;
}

int main() {
    std::vector<Point> points;
    std::vector<Spring> springs;
    std::vector<std::vector<Spring>> pointSprings;

    genPointsAndSprings(points, springs, pointSprings);
    Point *p_d;
    Spring **ps_d;
    std::vector<Spring *> psd;
    hipMalloc(&p_d, points.size() * sizeof(Point));
    hipMemcpy(p_d, &points[0], points.size() * sizeof(Point), hipMemcpyHostToDevice);
    hipMalloc(&ps_d, points.size() * sizeof(Spring *));
    for (int i = 0; i < points.size(); i++) {
    	Spring *s_d;
    	psd.push_back(s_d);
    	hipMalloc(&s_d, pointSprings[i].size() * sizeof(Spring));
    	hipMemcpy(s_d, &pointSprings[i], pointSprings[i].size() * sizeof(Spring), hipMemcpyHostToDevice);
    }

    double t = 0;
    // 60 fps - 0.000166
    double limit = 0.1;
  
  	int numSprings = (int)springs.size();
    printf("num springs evaluated: %lld\n", long long int(limit / dt * numSprings));
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    while (t < limit) {
        double adjust = 1 + sin(t * kOscillationFrequency) * 0.1;
        update_point<<<pointsPerSide * pointsPerSide, pointsPerSide>>>(p_d, ps_d, adjust);
 
        t += dt;
    }

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin);
    std::cout << "Time difference = " << ms.count() / 1000.0 << "[s]" << std::endl;

    Point *ps = (Point *)malloc(points.size() * sizeof(Point));
    hipMemcpy(ps, p_d, points.size() * sizeof(Point), hipMemcpyDeviceToHost);
    
    hipFree(p_d);
    for (int i = 0; i < points.size(); i++) {
    	hipFree(psd[i]);
    }
    hipFree(ps_d);
    free(ps);

    return 0;
}

void genPointsAndSprings(
	std::vector<Point> &points,
	std::vector<Spring> &springs,
	std::vector<std::vector<Spring>> &pointSprings) {
    std::map<int, std::map<int, std::map<int, Point>>> cache;

    for (int x = 0; x < pointsPerSide; x++) {
        for (int y = 0; y < pointsPerSide; y++) {
            for (int z = 0; z < pointsPerSide; z++) {
                // (0,0,0) or (0.1,0.1,0.1) and all combinations
                Point p = {x / 10.0, kDropHeight + y / 10.0, z / 10.0, 0, 0, 0, 0.1, 0, 0, 0, 0};
                points.push_back(p);
                if (cache.count(x) == 0) {
                    cache[x] = {};
                }
                if (cache[x].count(y) == 0) {
                    cache[x][y] = {};
                }
                cache[x][y][z] = p;
                pointSprings.push_back({});
            }
        }
    }
    // Create the springs
    for (int x = 0; x < pointsPerSide; x++) {
        for (int y = 0; y < pointsPerSide; y++) {
            for (int z = 0; z < pointsPerSide; z++) {
                Point p1 = cache[x][y][z];
                int p1index = z + pointsPerSide * y + pointsPerSide * pointsPerSide * x;
                for (int x1 = x; x1 < x + 2; x1++) {
                    if (x1 == pointsPerSide) {
                        continue;
                    }
                    for (int y1 = y; y1 < y + 2; y1++) {
                        if (y1 == pointsPerSide) {
                            continue;
                        }
                        for (int z1 = z; z1 < z + 2; z1++) {
                            if (z1 == pointsPerSide || (x1 == x && y1 == y && z1 == z)) {
                                continue;
                            }
                            Point p2 = cache[x1][y1][z1];
                            int p2index = z1 + pointsPerSide * y1 + pointsPerSide * pointsPerSide * x1;
                            double length = sqrt(pow(p1.x - p2.x, 2) + pow(p1.y - p2.y, 2) + pow(p1.z - p2.z, 2));
                            Spring s = {kSpring, p1index, p2index, length};
                            springs.push_back(s);
                            pointSprings[p1index].push_back(s);
							pointSprings[p2index].push_back(s);
							p2.numSprings += 1;
							p1.numSprings += 1;
                        }
                    }
                }
            }
        }
    }
}
